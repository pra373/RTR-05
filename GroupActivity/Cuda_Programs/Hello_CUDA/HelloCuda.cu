#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

//cuda headers
#include<hip/hip_runtime.h>

//global variables
const int iNmuberofArrayElements = 5;

float *HostInput1 = NULL;
float *HostInput2 = NULL;
float *HostOutput = NULL;

float *DeviceInput1 = NULL;
float *DeviceInput2 = NULL;
float *DeviceOutput = NULL;

//cuda Kernel

__global__ void VecAddGPU(float *in1, float *in2, float *out, int len)
{
	//code
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

//entry point function

int main(void)
{
	//function declaration
	void Cleanup(void);

	//variable declaration
	int size = iNmuberofArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	//code
	//host memory allocation

	HostInput1 = (float *)malloc(size);
	if (HostInput1 == NULL)
	{
		printf("Host memory allocation is failed for HostInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	HostInput2 = (float *)malloc(size);
	if (HostInput2 == NULL)
	{
		printf("Host memory allocation is failed for HostInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	HostOutput = (float *)malloc(size);

	if (HostOutput == NULL)
	{
		printf("Host memory allocation is failed for HostOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//filling values into host arrays
	HostInput1[0] = 101.0f;
	HostInput1[1] = 102.0f;
	HostInput1[2] = 103.0f;
	HostInput1[3] = 104.0f;
	HostInput1[4] = 105.0f;

	HostInput2[0] = 201.0f;
	HostInput2[1] = 202.0f;
	HostInput2[2] = 203.0f;
	HostInput2[3] = 204.0f;
	HostInput2[4] = 205.0f;

	//device memory allocation

	result = hipMalloc((void **)&DeviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceOutput, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host arrays into device arrays

	result = hipMemcpy(DeviceInput1, HostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data copy is failed for DeviceInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(DeviceInput2, HostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data copy is failed for DeviceInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	dim3 dimGrid = dim3(iNmuberofArrayElements, 1, 1);
	dim3 dimBlock = dim3(1, 1, 1);

	//cuda kernel for vector addition

	VecAddGPU << <dimGrid, dimBlock >> > (DeviceInput1, DeviceInput2, DeviceOutput, iNmuberofArrayElements);

	//copy data from device array into host array

	result = hipMemcpy(HostOutput, DeviceOutput, size, hipMemcpyDeviceToHost);

	if (result != hipSuccess)
	{
		printf("Device to host data copy is failed for HostOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}
	// vector addition on host 
	for (int i = 0; i < iNmuberofArrayElements; i++)
	{
		printf("%f + %f = %f\n", HostInput1[i], HostInput2[i], HostOutput[i]);
	}

	//cleanup
	Cleanup();

	return(0);









}

void Cleanup(void)
{
	//code
	if (DeviceOutput)
	{
		hipFree(DeviceOutput);
		DeviceOutput = NULL;
	}

	if (DeviceInput2)
	{
		hipFree(DeviceInput2);
		DeviceInput2 = NULL;
	}

	if (DeviceInput1)
	{
		hipFree(DeviceInput1);
		DeviceInput1 = NULL;
	}

	
}


