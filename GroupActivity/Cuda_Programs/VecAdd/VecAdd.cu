#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

//cuda headers
#include<hip/hip_runtime.h>
#include"helper_timer.h"

//global variables
const int iNumberOfArrayElements = 11444777;

float *HostInput1 = NULL;
float *HostInput2 = NULL;
float *HostOutput = NULL;

float *gold = NULL;

float *DeviceInput1 = NULL;
float *DeviceInput2 = NULL;
float *DeviceOutput = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

//cuda kernel

__global__ void VecAddGPU(float *in1, float *in2, float *out, int len)
{
	//code
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

//entry point function

int main(void)
{
	//function declaration
	void Cleanup(void);
	void fillFloatArrayWithRandomNumbers(float *, int);
	void VecAddCPU(const float *, const float *, float *, int);

	//variable declaration
	int size = iNumberOfArrayElements * sizeof(float);
	hipError_t result = hipSuccess;

	//code
	//host memory allocation

	HostInput1 = (float *)malloc(size);
	if (HostInput1 == NULL)
	{
		printf("Host memory allocation is failed for HostInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	HostInput2 = (float *)malloc(size);
	if (HostInput2 == NULL)
	{
		printf("Host memory allocation is failed for HostInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	HostOutput = (float *)malloc(size);

	if (HostOutput == NULL)
	{
		printf("Host memory allocation is failed for HostOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	gold = (float *)malloc(size);
	if (gold == NULL)
	{
		printf("Host memory allocation is failed for gold array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//filling values into host arrays
	fillFloatArrayWithRandomNumbers(HostInput1, iNumberOfArrayElements);
	fillFloatArrayWithRandomNumbers(HostInput2, iNumberOfArrayElements);

	//device memory allocation

	result = hipMalloc((void **)&DeviceInput1, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceInput2, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceOutput, size);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host arrays into device arrays

	result = hipMemcpy(DeviceInput1, HostInput1, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data copy is failed for DeviceInput1 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(DeviceInput2, HostInput2, size, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to device data copy is failed for DeviceInput2 array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//cuda kernel configuration

	dim3 dimGrid = dim3((int)ceil((float)iNumberOfArrayElements / 256.0f), 1, 1);
	dim3 dimBlock = dim3(256, 1, 1);

	//CUDA Kernel for vector addition

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	VecAddGPU << <dimGrid, dimBlock >> > (DeviceInput1, DeviceInput2, DeviceOutput, iNumberOfArrayElements);
	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

	//copy data from device array into host array

	result = hipMemcpy(HostOutput, DeviceOutput, size, hipMemcpyDeviceToHost);

	if (result != hipSuccess)
	{
		printf("Device to host data copy is failed for HostOutput array.\n");
		Cleanup();
		exit(EXIT_FAILURE);
	}

	//vector addition on host

	VecAddCPU(HostInput1, HostInput2, gold, iNumberOfArrayElements);

	//comparison

	const float epsilon = 0.000001f;
	int breakValue = -1;
	bool bAccuracy = true;

	for (int i = 0; i < iNumberOfArrayElements; i++)
	{
		float val1 = gold[i];
		float val2 = HostOutput[i];
		if (fabs(val1 - val2) > epsilon)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	char str[128];
	if (bAccuracy == false)
	{
		sprintf(str, "Comparison of CPU and GPU Vector Addition is not within accuracy of 0.000001 at array index %d\n", breakValue);

	}

	else
	{
		sprintf(str, "Comparison of CPU and GPU Vector Addition is within accuracy of 0.000001 at array index\n");

	}

	//output
	printf("\n\n");
	printf("Array1 begins from 0th index %.6f to %dth index %.6f\n\n", HostInput1[0], iNumberOfArrayElements - 1, HostInput1[iNumberOfArrayElements - 1]);
	printf("Array2 begins from 0th index %.6f to %dth index %.6f\n\n", HostInput2[0], iNumberOfArrayElements - 1, HostInput2[iNumberOfArrayElements - 1]);
	printf("Cuda Kernel Grid dimension = %d,%d,%d and Block dimension = %d,%d,%d\n\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

	printf("Output array begins from 0th index %.6f to %dth index %.6f\n\n", HostOutput[0], iNumberOfArrayElements - 1, HostOutput[iNumberOfArrayElements - 1]);
	printf("Time taken for vector addition on CPU = %.6f\n\n", timeOnCPU);
	printf("Time taken for vector addition on GPU = %.6f\n\n", timeOnGPU);
	printf("%s\n", str);

	//cleanup
	Cleanup();

	return(0);

}

void fillFloatArrayWithRandomNumbers(float *arr, int len)
{
	//code

	const float fscale = 1.0f / (float)RAND_MAX;
	for (int i = 0; i < len; i++)
	{
		arr[i] = fscale * rand();
	}

}
void VecAddCPU(const float *arr1, const float *arr2, float *out, int len)
{
	//code
	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0; i < len; i++)
	{
		out[i] = arr1[i] + arr2[i];
	}

	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;


	
	
}

void Cleanup(void)
{
	//code
	if (DeviceOutput)
	{
		hipFree(DeviceOutput);
		DeviceOutput = NULL;
	}

	if (DeviceInput2)
	{
		hipFree(DeviceInput2);
		DeviceInput2 = NULL;
	}

	if (DeviceInput1)
	{
		hipFree(DeviceInput1);
		DeviceInput1 = NULL;
	}

	if (gold)
	{
		hipFree(gold);
		gold = NULL;
	}

	if (HostOutput)
	{
		hipFree(HostOutput);
		HostOutput = NULL;
	}

	if (HostInput2)
	{
		hipFree(HostInput2);
		HostInput2 = NULL;
	}

	if (HostInput1)
	{
		hipFree(HostInput1);
		HostInput1 = NULL;
	}


}
