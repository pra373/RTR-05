#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

//cuda headers
#include<hip/hip_runtime.h>
#include"helper_timer.h"

//macros
#define BLOCK_WIDTH 128

//global variables

int *HostA = NULL;
int *HostB = NULL;
int *HostC = NULL;
int *Gold = NULL;

int *DeviceA = NULL;
int *DeviceB = NULL;
int *DeviceC = NULL;

float timeOnCPU = 0.0f;
float timeOnGPU = 0.0f;

//cuda kernel function

__global__ void matMulGPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
	//variable declaration
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int column = blockIdx.x * blockDim.x + threadIdx.x;

	//code

	if ((row < numARows) && (column < numBColumns))
	{
		int value = 0.0;
		for (int k = 0; k < numAColumns; k++)
		{
			int a = A[row * numAColumns + k];
			int b = B[k * numBColumns + column];
			value += a * b;
		}
		C[row * numCColumns + column] = value;
	}
}

int main(int argc, char *argv[])
{
	//function declarations
	void InitA(int *data, int, int);
	void InitB(int *data, int, int);
	void matMulCPU(int *, int *, int *, int, int, int, int);
	void cleanup(void);

	//variable declaration

	int numARows = BLOCK_WIDTH;
	int numAColumns = BLOCK_WIDTH;
	int numBRows = BLOCK_WIDTH;
	int numBColumns = BLOCK_WIDTH;

	int numCRows = numARows;
	int numCColumns = numBColumns;

	int numGoldRows = numARows;
	int numGoldColumns = numBColumns;

	int sizeA = numARows * numAColumns * sizeof(int);
	int sizeB = numBRows * numBColumns * sizeof(int);
	int sizeC = numCRows * numCColumns * sizeof(int);
	int sizeGold = numGoldRows * numGoldColumns * sizeof(int);

	hipError_t result = hipSuccess;

	//code
	//host memory allocation

	HostA = (int *)malloc(sizeA);
	if (HostA == NULL)
	{
		printf("Host memory allocation is failed for HostA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	HostB = (int *)malloc(sizeB);
	if (HostB == NULL)
	{
		printf("Host memory allocation is failed for HostB matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	HostC = (int *)malloc(sizeC);
	if (HostC == NULL)
	{
		printf("Host memory allocation is failed for HostC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	Gold = (int *)malloc(sizeGold);
	if (Gold == NULL)
	{
		printf("Host memory allocation is failed for Gold matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//printing matrix dimension and sizes

	printf("The Dimensions of matrix 'HostA' are : %d x %d \n", numARows, numAColumns);
	printf("The Dimensions of matrix 'HostB' are : %d x %d \n", numBRows, numBColumns);
	printf("The Dimensions of matrix 'HostC' are : %d x %d \n", numCRows, numCColumns);
	printf("The Dimensions of matrix 'Gold' are : %d x %d \n", numGoldRows, numGoldColumns);

	printf("Size of matrix HostA = %d\n", sizeA);
	printf("Size of matrix HostB = %d\n", sizeB);
	printf("Size of matrix HostC = %d\n", sizeC);
	printf("Size of matrix Gold = %d\n", sizeGold);

	//fill source matrices

	InitA(HostA, numARows, numAColumns);
	InitB(HostB, numBRows, numBColumns);

	//device memory allocation

	result = hipMalloc((void **)&DeviceA, sizeA);
	if(result!=hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceB, sizeB);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceB matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMalloc((void **)&DeviceC, sizeC);
	if (result != hipSuccess)
	{
		printf("Device memory allocation is failed for DeviceC matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//copy data from host matrices to device matrices

	result = hipMemcpy(DeviceA, HostA, sizeA, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for DeviceA matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	result = hipMemcpy(DeviceB, HostB, sizeB, hipMemcpyHostToDevice);
	if (result != hipSuccess)
	{
		printf("Host to Device data copy is failed for DeviceB matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//cuda kernel configuration

	dim3 dimGrid = dim3(ceil((int)numBColumns / (int)BLOCK_WIDTH), ceil((int)numARows / (int)BLOCK_WIDTH), 1);
	dim3 dimBlock = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	//cuda kernel for matrix multiplication

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	matMulGPU << <dimGrid, dimBlock >> > (DeviceA, DeviceB, DeviceC, numARows, numAColumns, numBColumns, numCColumns);

	sdkStopTimer(&timer);
	timeOnGPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

	//copy data from device matrices to host matrices
	
	result = hipMemcpy(HostC, DeviceC, sizeC, hipMemcpyDeviceToHost);

	if (result != hipSuccess)
	{
		printf("Device to Host data copy is failed for Hostc matrix.\n");
		cleanup();
		exit(EXIT_FAILURE);
	}

	//matrix multiplication on host

	matMulCPU(HostA, HostB, Gold, numARows, numAColumns, numBColumns, numCColumns);

	//comparison

	int breakValue = -1;
	bool bAccuracy = true;

	for (int i = 0; i < numCRows * numCColumns; i++)
	{
		int val1 = Gold[i];
		int val2 = HostC[i];
		if (val1 != val2)
		{
			bAccuracy = false;
			breakValue = i;
			break;
		}
	}

	char str[128];
	if (bAccuracy == false)
	{
		sprintf(str, "Comparison of CPU and GPU matrix multiplication is not accurate at array index %d.\n", breakValue);
	}

	else
	{
		sprintf(str, "Comparison of CPU and GPU matrix multiplication is accurate.\n");

	}

	printf("Time taken for matrix multiplication on CPU =%.6f\n", timeOnCPU);
	printf("Time taken for matrix multiplication on GPU =%.6f\n", timeOnGPU);

	printf("%s\n", str);

	//cleanup()

	cleanup();

	return(0);
}

void InitA(int *data, int row, int col)
{
	int num = 1;
	//code
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			*(data + i * col + j) = num;
			num++;
		}
	}
}

void InitB(int *data, int row, int col)
{
	int num = BLOCK_WIDTH;
	//code
	for (int i = 0; i < row; i++)
	{
		for (int j = 0; j < col; j++)
		{
			*(data + i * col + j) = num;
			num--;
		}
	}
}

void matMulCPU(int *A, int *B, int *C, int numARows, int numAColumns, int numBColumns, int numCColumns)
{
	//code

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkStartTimer(&timer);

	for (int i = 0; i < numARows; ++i)
	{
		for (int j = 0; j < numBColumns; ++j)
		{
			int value = 0.0f;
			for (int k = 0; k < numAColumns; ++k)
			{
				int a = A[i * numAColumns + k];
				int b = B[k * numBColumns + j];
				value += a * b;

			}

			C[i * numCColumns + j] = value;

		}
	}
	sdkStopTimer(&timer);
	timeOnCPU = sdkGetTimerValue(&timer);
	sdkDeleteTimer(&timer);
	timer = NULL;

}

void cleanup(void)
{
	if (DeviceC)
	{
		hipFree(DeviceC);
		DeviceC = NULL;
	}

	if (DeviceB)
	{
		hipFree(DeviceB);
		DeviceB = NULL;
	}

	if (DeviceA)
	{
		hipFree(DeviceA);
		DeviceA = NULL;
	}

	if (Gold)
	{
		free(Gold);
		Gold = NULL;
	}

	if (HostC)
	{
		free(HostC);
		HostC = NULL;
	}

	if (HostB)
	{
		free(HostB);
		HostB = NULL;
	}

	if (HostA)
	{
		free(HostA);
		HostA = NULL;
	}


}

