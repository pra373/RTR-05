#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void add(int num1, int num2, int *c)
{
	*c = num1 + num2;
}

int main(void)
{
	int result;
	int* dev_c;
	int a = 5;
	int b = 5;

	hipError_t error;

	// allocate memory on the device

	error = hipMalloc((void**)&dev_c, sizeof(int));

	if (error != hipSuccess)
	{
		printf("\nCuda Malloc Failed !!\n");
		exit(0);
	}

	// call the kernel

	add << <1, 1 >> > (a, b, dev_c);   // do addition on GPU and store the result on allocated memory block on device

	// copy the reslut from devices memory to host memory to bring the result in host code

	error = hipMemcpy(&result, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("\ncudaMemcpy() Failed !!!\n");
		exit(0);
	}

	printf("\n\n%d + %d = %d\n\n", a, b, result);

	hipFree(dev_c);



	return(0);
}