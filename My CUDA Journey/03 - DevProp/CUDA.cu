#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


int main(void)
{
	hipDeviceProp_t properties;

	int deviceCount;

	hipGetDeviceCount(&deviceCount);

	printf("\n\nTotal number of CUDA Devices: %d\n\n", deviceCount);

	hipGetDeviceProperties(&properties, 0);

	printf("\n\n ------ General Information for device 0 ------\n\n");

    // Print device properties
    printf("\nDevice 0: %s\n", properties.name);
	printf("Clock Rate: %d\n", properties.clockRate);
	printf("Compute capability: %d.%d\n", properties.major, properties.minor);
	printf("Multiprocessor count: %d\n",properties.multiProcessorCount);
	printf("Max threads per block: %d\n", properties.maxThreadsPerBlock);
	printf("Max Number of blocks in X Direction %d\n", properties.maxGridSize[0]);
	printf("Max Number of blocks in Y Direction %d\n", properties.maxGridSize[1]);
	printf("Max Number of blocks in Z Direction %d\n", properties.maxGridSize[2]);
	
	printf("\n\n --- Memory Information for device 0 ---\n\n");
	printf("Total global mem: %zu\n", properties.totalGlobalMem);
	printf("Total constant Mem: %zu\n", properties.totalConstMem);
	printf("Max mem pitch: %zu\n", properties.memPitch);
	printf("Texture Alignment: %zu\n", properties.textureAlignment);

   

    return 0;

	
}