#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<math.h>

#define N 4096

__global__ void vecAdd(int* a, int* b, int* c)
{
	int bid = 0;

	bid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (bid < N)
	{
		c[bid] = a[bid] + b[bid];
	}
}

int main(void)
{
	int a[N], b[N], c[N];  // allocate three arrays on host side first 2 to pass data to GPU and thied on to receive the result from the GPU

	int i = 0;
	unsigned int count = 0;

	
	int threadsPerBlock = 512;

	int blocks = (int)ceil(((float)(N) / (float)threadsPerBlock));

	int* dev_a, *dev_b, *dev_c;  // declaring 3 pointers to store device memory address after allocating memory on the device

	hipError_t error;  //to catch error from Cuda APIs

	for (i = 0; i < N; i++)   // fill arrays on CPU side ( host side )
	{
		a[i] = -i;
		b[i] = i + i;
		count += 1;
	}

	error = hipMalloc((void**)&dev_a, N * sizeof(int));

	if (error != hipSuccess)
	{
		printf("Memory Allocation for input array A on the device is failed !!!");
		exit(0);
	}

	error = hipMalloc((void**)&dev_b, N * sizeof(int));

	if (error != hipSuccess)
	{
		printf("Memory Allocation for input array B on the device is failed !!!");
		exit(0);
	}

	error = hipMalloc((void**)&dev_c, N * sizeof(int));

	if (error != hipSuccess)
	{
		printf("Memory Allocation for input array C on the device is failed !!!");
		exit(0);
	}

	// now as the memory is allocated successfully on the GPU we wll copy data from host side to device side

	error = hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("Copying memory from Host to device for array A is failed");
		exit(0);
	}

	error = hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("Copying memory from Host to device for array b is failed");
		exit(0);
	}

	// now as the memory is copied from host to device successfully
	//we can run the kernel on this two arrays and store the result on array pointed by dev_c on the device

	vecAdd << <blocks, threadsPerBlock >> > (dev_a, dev_b, dev_c);

	// now as thr result is stored on array pointed by dev_c on the gpu we will copy the array on the host side

	error = hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("Copying memory from device to host for result array (c) is failed");
		exit(0);
	}

	//display the result

	printf("\n\n");

	for (i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	printf("count = %d", count);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	dev_a = NULL;
	dev_b = NULL;
	dev_c = NULL;

	return(0);
}
