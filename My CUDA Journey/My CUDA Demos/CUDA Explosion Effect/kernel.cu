#include "hip/hip_runtime.h"
#pragma once

// Declare explodeFactor in constant memory

__global__ void updatePosition_GPU(float* pInteropBuffer,float* explodeFactor, int numTriangles)
{
    

	int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIndex < numTriangles)
	{
		// Calculate the base index for the triangle being processed

		int baseIndex = threadIndex * 9;

        // Fetch the vertices of the triangle
        float v1x = pInteropBuffer[baseIndex];       // x-coordinate of first vertex
        float v1y = pInteropBuffer[baseIndex + 1];   // y-coordinate of first vertex
        float v1z = pInteropBuffer[baseIndex + 2];   // z-coordinate of first vertex

        float v2x = pInteropBuffer[baseIndex + 3];   // x-coordinate of second vertex
        float v2y = pInteropBuffer[baseIndex + 4];   // y-coordinate of second vertex
        float v2z = pInteropBuffer[baseIndex + 5];   // z-coordinate of second vertex

        float v3x = pInteropBuffer[baseIndex + 6];   // x-coordinate of third vertex
        float v3y = pInteropBuffer[baseIndex + 7];   // y-coordinate of third vertex
        float v3z = pInteropBuffer[baseIndex + 8];   // z-coordinate of third vertex

        // Calculate the normal for the triangle

        float abx = v2x - v1x; // Vector AB X component
        float aby = v2y - v1y; // Vector AB Y component
        float abz = v2z - v1z; // Vector AB Z component

        float acx = v3x - v1x; // Vector AC X component
        float acy = v3y - v1y; // Vector AC Y component
        float acz = v3z - v1z; // Vector AC Z component

        // Calculate the normal vector using the cross product

        float normalX = aby * acz - abz * acy; // X component
        float normalY = abz * acx - abx * acz; // Y component
        float normalZ = abx * acy - aby * acx; // Z component

        // Normalize the normal vector
        float magnitude = sqrt(normalX * normalX + normalY * normalY + normalZ * normalZ);

        const float epsilon = 1.0E-6;

        if (magnitude > epsilon)
        {
            normalX /= magnitude;
            normalY /= magnitude;
            normalZ /= magnitude;
        }

        // Update the position of the vertices based on the explode factor

        //update first vertex

        pInteropBuffer[baseIndex] = pInteropBuffer[baseIndex] + explodeFactor[0] * normalX;
        pInteropBuffer[baseIndex + 1] = pInteropBuffer[baseIndex + 1] + explodeFactor[1] * normalY;
        pInteropBuffer[baseIndex + 2] = pInteropBuffer[baseIndex + 2] + explodeFactor[2] * normalZ;

        //update second vertex

        pInteropBuffer[baseIndex + 3] = pInteropBuffer[baseIndex + 3] +  explodeFactor[0] * normalX;  
        pInteropBuffer[baseIndex + 4] = pInteropBuffer[baseIndex + 4] + explodeFactor[1] * normalY;
        pInteropBuffer[baseIndex + 5] = pInteropBuffer[baseIndex + 5] + explodeFactor[2] * normalZ;

        //update third vertex

        pInteropBuffer[baseIndex + 6] = pInteropBuffer[baseIndex + 6] + explodeFactor[0] * normalX; 
        pInteropBuffer[baseIndex + 7] = pInteropBuffer[baseIndex + 7] + explodeFactor[1] * normalY;
        pInteropBuffer[baseIndex + 8] = pInteropBuffer[baseIndex + 8] + explodeFactor[2] * normalZ;

	}

}

void launchCUDAKernel(float* pInteropBuffer, float* explodeFactor, int numTriangles)
{
    dim3 blockDim(1024, 1, 1);

    int numBlocks = ceil(numTriangles / 1024);

    //int numBlocks = (numTriangles + 1024 - 1) / 1024;

    dim3 gridDim(numBlocks, 1, 1);

    updatePosition_GPU << < gridDim, blockDim >> > (pInteropBuffer, explodeFactor, numTriangles);

}

